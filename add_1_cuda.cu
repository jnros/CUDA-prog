
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
   for (int i = 0; i < n; i++)
      y[i] = x[i]+y[i];
}

int main(void)
{
   int N = 1<<20; //1M elements

   // Allocate unified memory
   float *x, *y;
   hipMallocManaged(&x, N*sizeof(float));
   hipMallocManaged(&y, N*sizeof(float));

   // Initialize x and y arrays on the host
   for (int i = 0; i< N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
   }

   // Run kernel on 1M elements on the GPU
   add<<<1, 1>>>(N, x, y);

   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();

   float maxError = 0.0f;
      for (int i = 0; i < N; i++)
         maxError = fmax(maxError, fabs(y[i]-3.0f));
      std::cout << "Max error: " << maxError << std::endl;

   // Free memory
   hipFree(x); 
   hipFree(y); 

   return 0;
}
